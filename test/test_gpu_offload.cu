
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void testGPU() {
    printf("This runs on the GPU!\n");
}

int main() {
    printf("This runs on the CPU!\n");

    testGPU<<<1, 1>>>();

    hipDeviceSynchronize();

    return (0);
}